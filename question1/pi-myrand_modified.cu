#include "hip/hip_runtime.h"
// Source: http://web.mit.edu/pocky/www/cudaworkshop/MonteCarlo/PiMyRandom.cu

// Written by Barry Wilkinson, UNC-Charlotte. PiMyRandom.cu  December 22, 2010.
//Derived somewhat from code developed by Patrick Rogers, UNC-C

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <omp.h>
#define TRIALS_PER_THREAD 4096
#define BLOCKS 256
#define THREADS 256
#define PI 3.1415926535  // known value of pi

__device__ float my_rand(unsigned int *seed) {
	unsigned long a = 16807;  // constants for random number generator
  unsigned long m = 2147483647;   // 2^31 - 1
	unsigned long x = (unsigned long) *seed;

	x = (a * x)%m;

	*seed = (unsigned int) x;

  return ((float)x)/m;
}
__device__ double my_rand_dp(unsigned int *seed) {
	unsigned long a = 16807;  // constants for random number generator
  unsigned long m = 2147483647;   // 2^31 - 1
	unsigned long x = (unsigned long) *seed;

	x = (a * x)%m;

	*seed = (unsigned int) x;

  return ((double)x)/m;
}

float my_rand_host(unsigned int *seed) {
	unsigned long a = 16807;  // constants for random number generator
  unsigned long m = 2147483647;   // 2^31 - 1
	unsigned long x = (unsigned long) *seed;

	x = (a * x)%m;

	*seed = (unsigned int) x;

  return ((float)x)/m;
}
double my_rand_host_dp(unsigned int *seed) {
	unsigned long a = 16807;  // constants for random number generator
  unsigned long m = 2147483647;   // 2^31 - 1
	unsigned long x = (unsigned long) *seed;

	x = (a * x)%m;

	*seed = (unsigned int) x;

  return ((double)x)/m;
}

__global__ void gpu_monte_carlo(float *estimate,long N) {
	unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
	int points_in_circle = 0;
	float x, y;

	unsigned int seed =  tid + 1;  // starting number in random sequence

	for(int i = 0; i < N; i++) {
		x = my_rand(&seed);
		y = my_rand(&seed);
		points_in_circle += (x*x + y*y <= 1.0f); // count if x & y is in the circle.
	}
	estimate[tid] = 4.0f * points_in_circle / (float) N; // return estimate of pi
}
__global__ void gpu_monte_carlo_dp(double *estimate,long N) {
	unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
	int points_in_circle = 0;
	double x, y;

	unsigned int seed =  tid + 1;  // starting number in random sequence

	for(int i = 0; i < N; i++) {
		x = my_rand_dp(&seed);
		y = my_rand_dp(&seed);
		points_in_circle += (x*x + y*y <= 1.0f); // count if x & y is in the circle.
	}
	estimate[tid] = 4.0f * points_in_circle / (double) N; // return estimate of pi
}
float host_monte_carlo(long trials) { // # of Trial points we take to calculate PI
	float x, y;
	long points_in_circle;
	for(long i = 0; i < trials; i++) {
		x = rand() / (float) RAND_MAX;
		y = rand() / (float) RAND_MAX;
		points_in_circle += (x*x + y*y <= 1.0f);
	}
	return 4.0f * points_in_circle / trials;
}
float host_monte_carlo_dp(long trials) { // # of Trial points we take to calculate PI
	float x, y;
	long points_in_circle;
	for(long i = 0; i < trials; i++) {
		x = rand() / (float) RAND_MAX;
		y = rand() / (float) RAND_MAX;
		points_in_circle += (x*x + y*y <= 1.0f);
	}
	return 4.0f * points_in_circle / trials;
}
// CPU parallel
float CPU_parallel(int threads,int N){
	omp_set_dynamic(0);
	omp_set_num_threads(threads);
	float points_in_circle[threads];//= 0.0;
	long long trials = N[n]*THREADS*BLOCKS;
	float x = 0.0,y = 0.0;

	#pragma omp parallel
	{
		int tid = omp_get_thread_num();
		unsigned int seed = tid+1;
		int thre = threads;//omp_get_num_threads();
		long	istart = (tid * trials)/thre;
		long 	iend = ((tid+1)* trials)/thre;
		points_in_circle[tid] = 0.0;
		for(long long i = istart; i <iend; i++) {
			x = my_rand_host(&seed);
			y = my_rand_host(&seed);
			points_in_circle[tid] += (x*x + y*y <= 1.0f);
		}
	}
	float circle_points =0.0;
	for(int k=0;k<threads;k++){
		circle_points += points_in_circle[k];
	}
	float pi_cpu_par = 4.0*circle_points/trials;

	return pi_cpu_par;
}
double CPU_parallel_dp(int threads,int N){
	omp_set_dynamic(0);
	omp_set_num_threads(threads);
	double points_in_circle[threads];//= 0.0;
	long long trials = N*THREADS*BLOCKS;
	double x = 0.0,y = 0.0;

	#pragma omp parallel
	{
		int tid = omp_get_thread_num();
		int thre = threads;//omp_get_num_threads();
		long	istart = (tid * trials)/thre;
		long 	iend = ((tid+1)* trials)/thre;
		points_in_circle[tid] = 0.0;
		for(long long i = istart; i <iend; i++) {
			x = my_rand_host_dp(&seed);
			y = my_rand_host_dp(&seed);
			points_in_circle[tid] += (x*x + y*y <= 1.0f);
		}
	}
	double circle_points =0.0;
	for(int k=0;k<threads;k++){
		circle_points += points_in_circle[k];
	}
	double pi_cpu_par = 4.0*circle_points/trials;

	return pi_cpu_par;
}

int main (int argc, char *argv[]) {
	int threads = 2;

	if(argc == 2){
		threads = atoi(argv[1]);
	}
	clock_t start, stop;
	float host[BLOCKS * THREADS];
	float *dev;
	double host_dp[BLOCKS * THREADS];
	double *dev_dp;
	int N[4] = {1,256,1024,4096};

	for(int n=0;n<4;n++){
		if(n>0){
		printf("=====================SINGLE PRECISION===============================\n");
		printf("\n" );
		printf("\n" );
		printf("# of trials per thread = %d, # of blocks = %d, # of threads/block = %d.\n", N[n],BLOCKS, THREADS);
		printf("\n" );
	}
	start = clock();

	hipMalloc((void **) &dev, BLOCKS * THREADS * sizeof(float)); // allocate device mem. for counts

	gpu_monte_carlo<<<BLOCKS, THREADS>>>(dev,N[n]);

	hipMemcpy(host, dev, BLOCKS * THREADS * sizeof(float), hipMemcpyDeviceToHost); // return results

	float pi_gpu=0.0;
	for(int i = 0; i < BLOCKS * THREADS; i++) {
		pi_gpu += host[i];
	}

	pi_gpu /= (BLOCKS * THREADS);

	stop = clock();

	if(n>0){
		printf("CUDA estimate of PI = %f [error of %f]\n", pi_gpu, pi_gpu - PI);
		printf("GPU pi calculated in %f ms.\n", 1000* (stop-start)/(float)CLOCKS_PER_SEC);
		printf("\n" );
	}
	start = clock();
	float pi_cpu = host_monte_carlo(BLOCKS * THREADS * N[n]);
	stop = clock();
	if(n>0){
		printf("CPU estimate of PI = %f [error of %f]\n", pi_cpu, pi_cpu - PI);
		printf("CPU pi calculated in %f ms.\n",1000* (stop-start)/(float)CLOCKS_PER_SEC);
		printf("\n" );
	}
	// PI calculated from CPU parallel computation
	start = clock();

	float pi_cpu_par = CPU_parallel(threads,N[n]);

	stop = clock();

	if(n>0){
		printf("CPU parallel estimate of PI = %f [error of %f]\n", pi_cpu_par, pi_cpu_par - PI);
		printf("CPU parallel pi calculated in %f ms.With Thread count of %i\n",1000* (stop-start)/(float)CLOCKS_PER_SEC, threads);
		printf("\n" );
	}
	if(n>0){
	printf("=====================DOUBLE PRECISION===============================\n");
	printf("\n" );
	printf("\n" );
	printf("# of trials per thread = %d, # of blocks = %d, # of threads/block = %d.\n", N[n],BLOCKS, THREADS);
	printf("\n" );
}
start = clock();

hipMalloc((void **) &dev_dp, BLOCKS * THREADS * sizeof(double)); // allocate device mem. for counts

gpu_monte_carlo_dp<<<BLOCKS, THREADS>>>(dev_dp,N[n]);

hipMemcpy(host_dp, dev_dp, BLOCKS * THREADS * sizeof(double), hipMemcpyDeviceToHost); // return results

double pi_gpu_dp=0.0;
for(int i = 0; i < BLOCKS * THREADS; i++) {
	pi_gpu_dp += host_dp[i];
}

pi_gpu_dp /= (BLOCKS * THREADS);

stop = clock();

if(n>0){
	printf("CUDA estimate of PI = %f [error of %f]\n", pi_gpu_dp, pi_gpu_dp - PI);
	printf("GPU pi calculated in %f ms.\n", 1000* (stop-start)/(double)CLOCKS_PER_SEC);
	printf("\n" );
}
start = clock();
double pi_cpu_dp = host_monte_carlo_dp(BLOCKS * THREADS * N[n]);
stop = clock();
if(n>0){
	printf("CPU estimate of PI = %f [error of %f]\n", pi_cpu_dp, pi_cpu_dp - PI);
	printf("CPU pi calculated in %f ms.\n",1000* (stop-start)/(float)CLOCKS_PER_SEC);
	printf("\n" );
}
// PI calculated from CPU parallel computation
start = clock();
double pi_cpu_par_dp = CPU_parallel_dp(threads,N[n]);
stop = clock();
if(n>0){
	printf("CPU parallel estimate of PI = %f [error of %f]\n", pi_cpu_par_dp, pi_cpu_par_dp - PI);
	printf("CPU parallel pi calculated in %f ms.With Thread count of %i\n",1000* (stop-start)/(float)CLOCKS_PER_SEC, threads);
	printf("\n" );
}
}
	return 0;
}

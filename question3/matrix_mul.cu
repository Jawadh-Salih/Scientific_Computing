#include <ctype.h>
#include <unistd.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <omp.h>
#include <hip/hip_runtime.h>


#define TILE_N 10

float ** generate_matrix_sp(float **M, unsigned long N){
  M = (float **)malloc(sizeof(float *)*N);
  for(long i=0; i < N; i++) M[i] =(float *) malloc(sizeof(float) * N);
  for(long i=0;i<N;i++){
    for(long j=0;j<N;j++){
      M[i][j] =(float)rand()/(float)RAND_MAX +1;
    }
  }
  return M;
}
double ** generate_matrix_dp(double **M, unsigned long N){
  M = (double **)malloc(sizeof(double *)*N);
  for(long i=0; i < N; i++) M[i] = (double*)malloc(sizeof(double) * N);

  for(long i=0;i<N;i++){
    for(long j=0;j<N;j++){
      M[i][j] = (double)rand()/(double)RAND_MAX +1;
    }
  }
  return M;
}

// Single Precision Implementation
float** matrix_mul_sp(float **M1,float **M2, unsigned long N){

  float ** C = (float **)malloc(sizeof(float *)*N);
  for(long i=0; i < N; i++) C[i] =(float*) malloc(sizeof(float) * N);
  for(long i=0;i<N;i++){
    for(long j=0;j<N;j++){
        C[i][j]=0.0;
        for(long k=0;k<N;k++){
          C[i][j] += M1[i][k]*M2[k][j];
        }
    }
  }

  return C;
}

// Double Precision Implementation
double ** matrix_mul_dp(double **V1,double **V2, unsigned long N){

  double ** C = (double **)malloc(sizeof(double *)*N);
  for(long i=0; i < N; i++) C[i] = (double*)malloc(sizeof(double) * N);

  for(long i=0;i<N;i++){
    for(long j=0;j<N;j++){
        for(long k=0;k<N;k++){
          C[i][j] += V1[i][k]*V2[k][j];
        }
    }
  }

  return C;
}

__global__ void matrix_mul_sp_kernel(float *A_dev,float *B_dev,float * C_dev, unsigned long N){

  // to store the element of matrix
  int row = blockIdx.y*blockDim.y+threadIdx.y; // row index for the matrices
  int col = blockIdx.x*blockDim.x+threadIdx.x; // column index for the matrices

  if ((row < N) && (col <N)){
    float C_value = 0.0;
    for(int k=0;k<N;k++){
        C_value += A_dev[row*N+k]*B_dev[k*N+col];
        C_dev[row*N+col] = C_value;
    }
  }
}
__global__ void matrix_mul_dp_kernel(double *A_dev,double *B_dev,double * C_dev, unsigned long N){

  // to store the element of matrix
  int row = blockIdx.y*blockDim.y+threadIdx.y; // row index for the matrices
  int col = blockIdx.x*blockDim.x+threadIdx.x; // column index for the matrices

  if ((row < N) && (col <N)){
    double C_value = 0.0;
    for(int k=0;k<N;k++){
        C_value += A_dev[row*N+k]*B_dev[k*N+col];
        C_dev[row*N+col] = C_value;
    }
  }
}
int main(int argc,char *argv[]){

  int N[3] = {600,1200,1800};
  float **Asp,**Bsp; // Single precision
  double **Adp,**Bdp; // double precision

  int threads =2;
  if(argc == 2)
    threads = atoi(argv[1]);

  // int select_precision = 0; //default to floating point.
  // int  computation = 0; //
  // int verify =0;
  double time_serial_sp=0.0;
  double time_parallel_sp=0.0;
  double time_cuda_sp=0.0;
  double time_serial_dp=0.0;
  double time_parallel_dp=0.0;
  double time_cuda_dp=0.0;

      printf("========================================================================\n");
      printf("====================Matrix Multiplication calculation===================\n");
      printf("========================================================================\n");

      printf("Thread Count is %i\n", threads );

  for(int n=0;n<3;n++){
    // For GPU
    size_t size_M;
    dim3 dimGrid(N[n]/TILE_N,N[n]/TILE_N,1);
    dim3 dimBlock(TILE_N,TILE_N,1);

    // random seed as a Pseudo Random Number generator.
    unsigned seed = (time(NULL));
    srand(seed);
    clock_t start, stop;


    printf("==============================Single Precision==========================\n");

    Asp = generate_matrix_sp(Asp,N[n]);
    Bsp = generate_matrix_sp(Bsp,N[n]);

    //Start of CPU serial computation
    start  = clock();

    float ** C_sp = matrix_mul_sp(Asp,Bsp,N[n]);

    stop = clock();
    //End of CPU serial computation.
    time_serial_sp = 1000.0 * (stop-start)/(double)CLOCKS_PER_SEC;
    printf("Time taken to execute CPU Serial program : %g ms \n", time_serial_sp );

    //Start of CPU parallel computation in OMP.
    float ** C_sp_par = (float **)malloc(sizeof(float *)*N[n]);
    for(long i=0; i < N[n]; i++) C_sp_par[i] = (float *) malloc(sizeof(float) * N[n]);
    float dot_product_sp=0.0;

    start= clock();

    omp_set_num_threads(threads);
    #pragma omp parallel
    {
      int tid = omp_get_thread_num();
      int	istart = (tid * N[n])/threads;
      int 	iend = ((tid+1)* N[n])/threads;
      for(int i=istart;i<iend;i++){
        for(int j=0;j<N[n];j++){
          C_sp_par[i][j] =0.0;
          dot_product_sp=0.0;
          #pragma omp parallel for schedule(static) reduction(+:dot_product_sp)
          for(int k=0;k<N[n];k++){
            dot_product_sp += Asp[i][k]*Bsp[k][j];
          }
          C_sp_par[i][j] = dot_product_sp;
        }
      }
    }
    stop = clock();
    // End of CPU parallel Computation
    time_parallel_sp = 1000.0* (stop-start)/(double)CLOCKS_PER_SEC;
    printf("Time taken to execute CPU Parallel program : %g ms\n", time_parallel_sp);



    //Start of GPU computation.

    float *Csp_host, *Csp_dev;
    float *Asp_dev,*Bsp_dev; // Single precision
    size_M = N[n]*N[n]*sizeof(float);
    float *Asp_tmp = (float *)malloc(size_M),*Bsp_tmp= (float *)malloc(size_M);
    for(int l = 0;l<N[n];l++){
      for(int m=0;m<N[n];m++){
        Asp_tmp[l*N[n]+m] = Asp[l][m];
        Bsp_tmp[l*N[n]+m] = Bsp[l][m];
      }
    }


    start = clock();

    Csp_host = (float *)malloc(size_M);
    hipMalloc((void **) &Csp_dev, size_M);  // Allocate array on device
    hipMalloc((void **) &Asp_dev, size_M);  // Allocate array on device
    hipMalloc((void **) &Bsp_dev, size_M);  // Allocate array on device
    // Initialize array in device to 0
    hipMemset(Csp_dev, 0, size_M);
    hipMemcpy(Asp_dev, Asp_tmp, size_M, hipMemcpyHostToDevice);
    hipMemcpy(Bsp_dev, Bsp_tmp, size_M, hipMemcpyHostToDevice);

    matrix_mul_sp_kernel<<<dimGrid,dimBlock>>>(Asp_dev,Bsp_dev,Csp_dev,N[n]);

    hipMemcpy(Csp_host,Csp_dev, size_M, hipMemcpyDeviceToHost);

    hipFree(Csp_dev);hipFree(Asp_dev);hipFree(Bsp_dev);


    stop = clock();
    //End of GPU computation.
    time_cuda_sp = 1000.0*(stop-start)/(double)CLOCKS_PER_SEC;
    printf("Time taken to execute GPU Parallel program : %g ms\n", time_cuda_sp);



    printf("================================Double Precision========================\n");
    Adp = generate_matrix_dp(Adp,N[n]);
    Bdp = generate_matrix_dp(Bdp,N[n]);

    start = clock();
    double **C_dp =matrix_mul_dp(Adp,Bdp,N[n]);
    stop = clock();

    //End of CPU serial computation.
    time_serial_dp = 1000.0 * (stop-start)/(double)CLOCKS_PER_SEC;
    printf("Time taken to execute CPU Serial program : %g ms \n", time_serial_dp );


    //Start of CPU parallel computation in OMP
    double ** C_dp_par = (double **)malloc(sizeof(double *)*N[n]);
    for(long i=0; i < N[n]; i++) C_dp_par[i] =(double *) malloc(sizeof(double) * N[n]);

    double dot_product_dp = 0.0;

    start= clock();
    omp_set_num_threads(threads);
    #pragma omp parallel
    {
      int tid = omp_get_thread_num();
      int	istart = (tid * N[n])/threads;
      int 	iend = ((tid+1)* N[n])/threads;
      for(int i=istart;i<iend;i++){
        for(int j=0;j<N[n];j++){
          C_dp_par[i][j] =0.0;
          dot_product_dp=0.0;
          #pragma omp parallel for schedule(static) reduction(+:dot_product_dp)
          for(int k=0;k<N[n];k++){
            dot_product_dp += Adp[i][k]*Bdp[k][j];
          }
          C_dp_par[i][j] = dot_product_dp;
        }
      }
    }
    stop = clock();

    // End of CPU parallel computation.
    time_parallel_dp = 1000.0* (stop-start)/(double)CLOCKS_PER_SEC;
    printf("Time taken to execute CPU Parallel program  : %g ms \n", time_parallel_dp);


    //Start of GPU computation in CUDA
    double *Cdp_host, *Cdp_dev;
    double *Adp_dev,*Bdp_dev; // double precision
    size_M = N[n]*N[n]*sizeof(double);
    double *Adp_tmp = (double *)malloc(size_M),*Bdp_tmp= (double *)malloc(size_M);
    for(int l = 0;l<N[n];l++){
      for(int m=0;m<N[n];m++){
        Adp_tmp[l*N[n]+m] = Adp[l][m];
        Bdp_tmp[l*N[n]+m] = Bdp[l][m];
      }
    }

    start = clock();
    Cdp_host = (double *)malloc(size_M);
    hipMalloc((void **) &Cdp_dev, size_M);  // Allocate array on device
    hipMalloc((void **) &Adp_dev, size_M);  // Allocate array on device
    hipMalloc((void **) &Bdp_dev, size_M);  // Allocate array on device
    // Initialize array in device to 0
    hipMemset(Cdp_dev, 0, size_M);
    hipMemcpy(Adp_dev, Adp_tmp, size_M, hipMemcpyHostToDevice);
    hipMemcpy(Bdp_dev, Bdp_tmp, size_M, hipMemcpyHostToDevice);

    matrix_mul_dp_kernel<<<dimGrid,dimBlock>>>(Adp_dev,Bdp_dev,Cdp_dev,N[n]);

    hipMemcpy(Cdp_host,Cdp_dev, size_M, hipMemcpyDeviceToHost);

    hipFree(Cdp_dev);hipFree(Adp_dev);hipFree(Bdp_dev);
    stop = clock();
    //End of GPU computation.
    time_cuda_dp = 1000.0*(stop-start)/(double)CLOCKS_PER_SEC;

    printf("Time taken to execute GPU Parallel program  : %g ms \n", time_cuda_dp);

  }

}
// 2D Layout
// printf("\n" );
// for(long i=0;i<N;i++){
//   for(long j=0;j<N;j++){
//     printf("%lf ",C_sp_par[i][j]);
//   }
//   printf("\n" );
// }

// 1 D Layout
// int count =1;
// for(long i=0;i<N*N;i++){
//   printf("%f ",Csp_host[i]);
//   if(i== count*N-1){
//       printf("\n" );
//       count ++;
//   }
// }
//
//   printf("\n" );  printf("\n" );
